
#include <hip/hip_runtime.h>
#include <cstdint>
#include <memory.h>
#include <cstdio>
#include <ctime>
#include <thread>
#include <vector>
#include <mutex>
#include <chrono>
#include <string>

#define RANDOM_MULTIPLIER 0x5DEECE66DULL
#define RANDOM_ADDEND 0xBULL
#define RANDOM_MASK ((1ULL << 48ULL) - 1ULL)

#ifndef FLOOR_LEVEL
#define FLOOR_LEVEL 63LL
#endif

#ifndef WANTED_CACTUS_HEIGHT
#define WANTED_CACTUS_HEIGHT 10LL
#endif

#ifndef WORK_UNIT_SIZE
#define WORK_UNIT_SIZE (1ULL << 23ULL)
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256ULL
#endif

#ifndef GPU_COUNT
#define GPU_COUNT 1ULL
#endif

#ifndef OFFSET
#define OFFSET 0
#endif

#ifndef END
#define END (1ULL << 48ULL)
#endif

namespace java_random {

    // Random::next(bits)
    __device__ inline uint32_t next(uint64_t *random, int32_t bits) {
        *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
        return (uint32_t) (*random >> (48ULL - bits));
    }

    __device__ inline int32_t next_int_unknown(uint64_t *seed, int16_t bound) {
        if ((bound & -bound) == bound) {
            *seed = (*seed * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
            return (int32_t) ((bound * (*seed >> 17ULL)) >> 31ULL);
        }

        int32_t bits, value;
        do {
            *seed = (*seed * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
            bits = *seed >> 17ULL;
            value = bits % bound;
        } while (bits - value + (bound - 1) < 0);
        return value;
    }

    // Random::nextInt(bound)
    __device__ inline uint32_t next_int(uint64_t *random) {
        return java_random::next(random, 31) % 3;
    }

}

__global__ __launch_bounds__(256, 2) void crack(uint64_t seed_offset, int32_t *num_seeds, uint64_t *seeds) {
    uint64_t originalSeed = blockIdx.x * blockDim.x + threadIdx.x + seed_offset;
    uint64_t seed = originalSeed;

    int8_t heightMap[1024];

#pragma unroll
    for (int i = 0; i < 1024; i++) {
        heightMap[i] = FLOOR_LEVEL;
    }

    int8_t *currentHighestPos = heightMap, *posMap;
    int16_t initialPosX, initialPosY, initialPosZ;
    int16_t posX, posY, posZ;

    int16_t i, a, j;

    for (i = -90; i < 0; i += 9) {
        // Keep, most threads finish early this way
        if (*currentHighestPos - WANTED_CACTUS_HEIGHT - FLOOR_LEVEL < i)
            return;

        initialPosX = java_random::next(&seed, 4) + 8;
        initialPosZ = java_random::next(&seed, 4) + 8;

        initialPosY = java_random::next_int_unknown(&seed, (heightMap[initialPosX + initialPosZ * 32] + 1) * 2);

        for (a = 0; a < 10; a++) {
            posX = initialPosX + java_random::next(&seed, 3) - java_random::next(&seed, 3);
            posY = initialPosY + java_random::next(&seed, 2) - java_random::next(&seed, 2);
            posZ = initialPosZ + java_random::next(&seed, 3) - java_random::next(&seed, 3);

            posMap = heightMap + posX + posZ * 32;

            // Keep
            if (posY <= *posMap)
                continue;

            for (j = 0; j < 1 + java_random::next_int_unknown(&seed, java_random::next_int(&seed) + 1); j++) {
                if ((posY + j - 1) > *posMap || posY < 0) continue;
                if ((posY + j) <= heightMap[(posX + 1) + posZ * 32]) continue;
                if ((posY + j) <= heightMap[posX + (posZ - 1) * 32]) continue;
                if ((posY + j) <= heightMap[(posX - 1) + posZ * 32]) continue;
                if ((posY + j) <= heightMap[posX + (posZ + 1) * 32]) continue;

                heightMap[posX + posZ * 32]++;

                if (*currentHighestPos < *posMap) {
                    currentHighestPos = posMap;
                }
            }
        }

        if (*currentHighestPos - FLOOR_LEVEL >= WANTED_CACTUS_HEIGHT) {
            seeds[atomicAdd(num_seeds, 1)] = originalSeed;
            return;
        }
    }
}

struct GPU_Node {
    int* num_seeds;
    uint64_t* seeds;
};

void setup_gpu_node(GPU_Node* node, int32_t gpu) {
    hipSetDevice(gpu);
    hipMallocManaged(&node->num_seeds, sizeof(*node->num_seeds));
    hipMallocManaged(&node->seeds, 1ULL << 10ULL); // approx 1kb
}

GPU_Node nodes[GPU_COUNT];
uint64_t offset = OFFSET;
uint64_t count = 0;
std::mutex info_lock;

void gpu_manager(int32_t gpu_index) {
    std::string fileName = "kaktoos_seeds" + std::to_string(gpu_index) + ".txt";
    FILE *out_file = fopen(fileName.c_str(), "a");
    hipSetDevice(gpu_index);
    while (offset < END) {
        *nodes[gpu_index].num_seeds = 0;
        crack<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE, 0>>> (offset, nodes[gpu_index].num_seeds, nodes[gpu_index].seeds);
        info_lock.lock();
        offset += WORK_UNIT_SIZE;
        info_lock.unlock();
        hipDeviceSynchronize();
        for (int32_t i = 0, e = *nodes[gpu_index].num_seeds; i < e; i++) {
            fprintf(out_file, "%lld\n", (long long int)nodes[gpu_index].seeds[i]);
            printf("Found seed: %lld\n", (long long int)nodes[gpu_index].seeds[i]);
        }
        fflush(out_file);
        info_lock.lock();
        count += *nodes[gpu_index].num_seeds;
        info_lock.unlock();
    }
    fclose(out_file);
}

int main() {
    printf("Searching %ld total seeds...\n", END - OFFSET);

    std::thread threads[GPU_COUNT];

    time_t startTime = time(nullptr), currentTime;
    for(int32_t i = 0; i < GPU_COUNT; i++) {
        setup_gpu_node(&nodes[i], i);
        threads[i] = std::thread(gpu_manager, i);
    }

    using namespace std::chrono_literals;

    while (offset < END) {
        time(&currentTime);
        int timeElapsed = (int)(currentTime - startTime);
        double speed = (double)(offset - OFFSET) / (double)timeElapsed / 1000000.0;
        printf("Searched %lld seeds, offset: %lld found %lld matches. Time elapsed: %ds. Speed: %.2fm seeds/s. %f%%\n",
               (long long int)(offset - OFFSET),
               (long long int)offset,
               (long long int)count,
               timeElapsed,
               speed,
               (double)(offset - OFFSET) / (END - OFFSET) * 100);

        std::this_thread::sleep_for(0.5s);
    }

    for (auto &thread : threads) {
        thread.join();
    }

    printf("Done!\n");
    printf("But, verily, it be the nature of dreams to end.\n");

}